#include "hip/hip_runtime.h"
#include <jni/jni.hpp>

#include <iostream>

//#define N 2048 * 2048 // Number of elements in each vector

__global__ void cuda_saxpy(int N, float x, float * a, float * b, float * c)
{
  // Determine our unique global thread ID, so we know which element to process
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  
  for (int i = tid; i < N; i += stride)
    c[i] = x * a[i] + b[i];
}


extern "C" JNIEXPORT jint JNICALL JNI_OnLoad(JavaVM* vm, void*)
   {
    struct CudaCalculator
       {
        static constexpr auto Name() { return "CudaCalculator"; }

        CudaCalculator(JNIEnv&) { std::cout << "CUDA Native peer initialized" << std::endl; }
        CudaCalculator(const CudaCalculator&) = delete; // noncopyable
        ~CudaCalculator() { std::cout << "Native peer finalized" << std::endl; }

        jni::jint saxpy(
            jni::JNIEnv &env, 
            jni::jint N,
            jni::jfloat x,
            jni::Array<jni::jfloat> &ja,
            jni::Array<jni::jfloat> &jb, 
            jni::Array<jni::jfloat> &jc) {
                //jni::jsize na = a.Length(env);
                //jni::jfloat a0 = a.Get(env, 0);
                int deviceId;
                int numberOfSMs;
               
                hipGetDevice(&deviceId);
                hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);
                
                float *a, *b, *c;
                int size = N * sizeof (float);
                
                hipMallocManaged(&a, size);
                hipMallocManaged(&b, size);
                hipMallocManaged(&c, size);
                
                for(int i=0; i<N; i++) {
                    a[i] = ja.Get(env, i);
                    b[i] = jb.Get(env, i);
                    c[i] = jc.Get(env, i);
                }
                

                hipMemPrefetchAsync(a, size, deviceId);
                hipMemPrefetchAsync(b, size, deviceId);
                hipMemPrefetchAsync(c, size, deviceId);
            
                int threads_per_block = 256;
                int number_of_blocks = numberOfSMs * 32;
            
                cuda_saxpy <<<number_of_blocks, threads_per_block>>>( N, x, a, b, c );
                hipDeviceSynchronize(); // Wait for the GPU to finish
                
                // Print out the first and last 5 values of c for a quality check
                for( int i = 0; i < 5; ++i )
                  printf("c[%d] = %f, ", i, c[i]);
                printf ("\n");
                for( int i = N-5; i < N; ++i )
                  printf("c[%d] = %f, ", i, c[i]);
                printf ("\n");
            
                // Free all our allocated memory
                hipFree( a ); hipFree( b ); hipFree( c );
                
                return 0;
            }
       };

    jni::JNIEnv& env { jni::GetEnv(*vm) };

    #define METHOD(MethodPtr, name) jni::MakeNativePeerMethod<decltype(MethodPtr), (MethodPtr)>(name)

    jni::RegisterNativePeer<CudaCalculator>(env, jni::Class<CudaCalculator>::Find(env), "peer",
        jni::MakePeer<CudaCalculator>,
        "initialize",
        "finalize",
        METHOD(&CudaCalculator::saxpy, "saxpy") );

    return jni::Unwrap(jni::jni_version_1_2);
   }
